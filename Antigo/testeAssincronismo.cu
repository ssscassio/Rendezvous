#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>

/*
http://on-demand.gputechconf.com/gtc/2014/presentations/S4158-cuda-streams-best-practices-common-pitfalls.pdf

http://www.nvidia.com/content/gtc/documents/1122_gtc09.pdf


Possibilidade de utilizar CudaEvents para sinalizar a finalizar de uma serie de operações ou o espaço no buffer sendo utilizado.

**/
__global__ void somarVetores(double *A[], double*B[], double*C[]){

  int i = threadIdx.x;
  int j = threadIdx.y;

  C[i][j]=A[i][j]+B[i][j];
}

int main(){


  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  double  A_h[10][10],B_h[10][10],C_h[10][10]; //Variaveis no host
  double  *A_d,*B_d,*C_d; //Variaveis do Device

  //Inicializando vetores
  for(int i = 0; i <10; i++){
    for(int j = 0; j <10; j++){
      A_h[i][j] = i*j;
      B_h[i][j] = i*j+1;
    }
  }

  dim3 threadsPerBlock(10,10);


  size_t size = 10*10*sizeof(double);

  hipMalloc((void**)&A_d, size);
  hipMalloc((void**)&B_d, size);
  hipMalloc((void**)&C_d, size);

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float gpu_time = 0.0f;

  hipEventRecord(start, 0);
  hipMemcpyAsync(&A_d, &A_h, size, hipMemcpyHostToDevice,0);
  hipMemcpyAsync(&B_d, &B_h, size, hipMemcpyHostToDevice,0);
  hipMemcpyAsync(&C_d, &C_h, size, hipMemcpyHostToDevice,0);
  somarVetores<<<1,threadsPerBlock,0,0>>>(A_d,B_d,C_d);
  hipMemcpyAsync(&C_h, &C_h, size, hipMemcpyDeviceToHost,0);
  hipEventRecord(stop, 0);

  while (hipEventQuery(stop) == hipErrorNotReady)
     {
         counter++;
     }

  // print the cpu and gpu times
  printf("time spent executing by the GPU: %.2f\n", gpu_time);
  printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter);

      // release resources
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipHostFree(A_h);
  hipHostFree(B_h);
  hipHostFree(C_h);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);




}
